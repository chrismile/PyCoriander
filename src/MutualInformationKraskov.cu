#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2020-2023, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

extern "C" {

typedef unsigned uint;
typedef unsigned uint32_t;
typedef uint3 uvec3;
typedef float2 vec2;

/*
 * For more details, please refer to:
 * - https://journals.aps.org/pre/abstract/10.1103/PhysRevE.69.066138
 *
 * Global defines:
 * - MEMBER_COUNT: Number of entries to compute the MI for.
 * - k: Number of neighbors used in search.
 * - MAX_STACK_SIZE_BUILD: 2 * uint32_t(ceil(log(MEMBER_COUNT + 1))); 2*10 for 1000 entries.
 * - MAX_STACK_SIZE_KN: uint32_t(ceil(log(MEMBER_COUNT + 1))); 10 for 1000 entries.
 */

const float EPSILON = 1e-6;
const float EPSILON_NOISE = 1e-5;

#define KRASKOV_USE_RANDOM_NOISE

#ifdef KRASKOV_USE_RANDOM_NOISE
__device__ float getRandomFloatNorm(uvec3& rngState) {
    rngState.x ^= rngState.x << 16;
    rngState.x ^= rngState.x >> 5;
    rngState.x ^= rngState.x << 1;

    uint t = rngState.x;
    rngState.x = rngState.y;
    rngState.y = rngState.z;
    rngState.z = t ^ rngState.x ^ rngState.y;

    return rngState.z / float(4294967295u) * 2.0 - 1.0;
}
#endif


/**
 * Lanczos approximation of digamma function using weights by Viktor T. Toth.
 * - digamma = d/dx ln(Gamma(x)) = Gamma'(x) / Gamma(x) (https://en.wikipedia.org/wiki/Digamma_function)
 * - Lanczos approximation: https://www.rskey.org/CMS/index.php/the-library/11
 * - Weights: https://www.rskey.org/CMS/index.php/the-library/11
 *
 * This function could be extended for values < 1 by:
 * - float z = 1 - iz;
 * - if (iz < 1) return digammaValue - M_PI * cosf(M_PI * iz) / sinf(M_PI * iz);
 */
#define G (5.15f)
#define P0 (2.50662827563479526904f)
#define P1 (225.525584619175212544f)
#define P2 (-268.295973841304927459f)
#define P3 (80.9030806934622512966f)
#define P4 (-5.00757863970517583837f)
#define P5 (0.0114684895434781459556f)
__device__ float digamma(uint iz) {
    if (iz == 1u) {
        return -0.57721566490153287f;
    }
    float z = float(iz);
    float zh = z - 0.5f;
    float z1 = z + 1.0f;
    float z2 = z + 2.0f;
    float z3 = z + 3.0f;
    float z4 = z + 4.0f;
    float ZP = P0 + P1 / z + P2 / z1 + P3 / z2 + P4 / z3 + P5 / z4;
    float dZP = P1 / (z * z) + P2 / (z1 * z1) + P3 / (z2 * z2) + P4 / (z3 * z3) + P5 / (z4 * z4);
    float digammaValue = logf(zh + G) + zh / (zh + G) - dZP / ZP - 1.0f;
    return digammaValue;
}


// ----------------------------------------------------------------------------------
/*
 * Sort2D.
 */
__device__ void swapElements2D(float* referenceValues, float* queryValues, uint i, uint j) {
    float temp = referenceValues[i];
    referenceValues[i] = referenceValues[j];
    referenceValues[j] = temp;
    temp = queryValues[i];
    queryValues[i] = queryValues[j];
    queryValues[j] = temp;
}

__device__ void heapify2D(
        float* referenceValues, float* queryValues, float* valuesAxis,
        uint startIdx, uint i, uint numElements) {
    uint child;
    float childValue0, childValue1, arrayI;
    while ((child = 2 * i + 1) < numElements) {
        // Is left or right child larger?
        childValue0 = valuesAxis[startIdx + child];
        childValue1 = valuesAxis[startIdx + child + 1];
        if (child + 1 < numElements && childValue0 < childValue1) {
            childValue0 = childValue1;
            child++;
        }
        // Swap with child if it is larger than the parent.
        arrayI = valuesAxis[startIdx + i];
        if (arrayI >= childValue0) {
            break;
        }
        swapElements2D(referenceValues, queryValues, startIdx + i, startIdx + child);
        i = child;
    }
}

__device__ void heapSort2D(
        float* referenceValues, float* queryValues, float* valuesAxis,
        uint startIdx, uint endIdx) {
    uint numElements = endIdx - startIdx;

    // We can't use "i >= 0" with uint, thus adapt range and subtract 1 from i.
    uint i;
    for (i = numElements / 2; i > 0; i--) {
        heapify2D(referenceValues, queryValues, valuesAxis, startIdx, i - 1, numElements);
    }
    // Largest element is at index 0. Swap it to the end of the processed array portion iteratively.
    for (i = 1; i < numElements; i++) {
        swapElements2D(referenceValues, queryValues, startIdx, startIdx + numElements - i);
        heapify2D(referenceValues, queryValues, valuesAxis, startIdx, 0, numElements - i);
    }
}
// ----------------------------------------------------------------------------------


// ----------------------------------------------------------------------------------
/*
 * Sort1D.
 */
__device__ void swapElements(
        float* kthNeighborDistances, float* valueArray, uint i, uint j) {
    float temp = valueArray[i];
    valueArray[i] = valueArray[j];
    valueArray[j] = temp;
    temp = kthNeighborDistances[i];
    kthNeighborDistances[i] = kthNeighborDistances[j];
    kthNeighborDistances[j] = temp;
}

__device__ void heapify(
        float* kthNeighborDistances, float* valueArray,
        uint i, uint numElements) {
    uint child;
    float childValue0, childValue1;
    while ((child = 2 * i + 1) < numElements) {
        // Is left or right child larger?
        childValue0 = valueArray[child];
        childValue1 = valueArray[child + 1];
        if (child + 1 < numElements && childValue0 < childValue1) {
            childValue0 = childValue1;
            child++;
        }
        // Swap with child if it is larger than the parent.
        if (valueArray[i] >= childValue0) {
            break;
        }
        swapElements(kthNeighborDistances, valueArray, i, child);
        i = child;
    }
}

__device__ void heapSort(float* kthNeighborDistances, float* valueArray) {
    // We can't use "i >= 0" with uint, thus adapt range and subtract 1 from i.
    uint i;
    for (i = MEMBER_COUNT / 2; i > 0; i--) {
        heapify(kthNeighborDistances, valueArray, i - 1, MEMBER_COUNT);
    }
    // Largest element is at index 0. Swap it to the end of the processed array portion iteratively.
    for (i = 1; i < MEMBER_COUNT; i++) {
        swapElements(kthNeighborDistances, valueArray, 0, MEMBER_COUNT - i);
        heapify(kthNeighborDistances, valueArray, 0, MEMBER_COUNT - i);
    }
}

__device__ float averageDigamma(float* kthNeighborDistances, float* valueArray) {
    heapSort(kthNeighborDistances, valueArray);
    float factor = 1.0 / float(MEMBER_COUNT);
    float meanDigammaValue = 0.0;
    for (uint c = 0; c < MEMBER_COUNT; c++) {
        float kthDist = kthNeighborDistances[c] - EPSILON;
        float currentValue = valueArray[c];
        float searchValueLower = currentValue - kthDist;
        float searchValueUpper = currentValue + kthDist;
        int lower = 0;
        int upper = MEMBER_COUNT;
        int middle = 0;
        // Binary search.
        while (lower < upper) {
            middle = (lower + upper) / 2;
            float middleValue = valueArray[middle];
            if (middleValue < searchValueLower) {
                lower = middle + 1;
            } else {
                upper = middle;
            }
        }

        int startRange = upper;
        lower = startRange;
        upper = MEMBER_COUNT;

        // Binary search.
        while (lower < upper) {
            middle = (lower + upper) / 2;
            float middleValue = valueArray[middle];
            if (middleValue < searchValueUpper) {
                lower = middle + 1;
            } else {
                upper = middle;
            }
        }
        int endRange = upper - 1;

        uint numPoints = max(uint(endRange + 1 - startRange), 1u);
        meanDigammaValue += factor * digamma(numPoints);
    }
    return meanDigammaValue;
}
// ----------------------------------------------------------------------------------


// ----------------------------------------------------------------------------------
/*
 * k-d tree.
 */
const uint INVALID_NODE = 0xFFFFu;
#define FLT_MAX 3.402823466e+38

struct KdNode {
    __device__ KdNode() {}
    __device__ KdNode(float x, float y, uint axis, uint leftRightIdx) : axis(axis), leftRightIdx(leftRightIdx) {
        this->point[0] = x;
        this->point[1] = y;
    }
    float point[2];
    uint axis;
    uint leftRightIdx;
};

struct StackEntryBuild {
    __device__ StackEntryBuild() {}
    __device__ StackEntryBuild(uint startIdx, uint endIdx, uint depth) : startIdx(startIdx), endIdx(endIdx), depth(depth) {}
    uint startIdx;
    uint endIdx;
    uint depth;
};

__device__ void buildKdTree(KdNode* nodes, float* referenceValues, float* queryValues) {
    uint nodeCounter = 0;
    StackEntryBuild stack[MAX_STACK_SIZE_BUILD];
    uint stackSize = 1u;
    stack[0] = StackEntryBuild(0u, MEMBER_COUNT, 0u);
    StackEntryBuild stackEntry;
    while (stackSize > 0u) {
        stackSize--;
        stackEntry = stack[stackSize];

        uint axis = stackEntry.depth % 2u;
        heapSort2D(
                referenceValues, queryValues, axis == 0u ? referenceValues : queryValues,
                stackEntry.startIdx, stackEntry.endIdx);
        uint medianIndex = stackEntry.startIdx + (stackEntry.endIdx - stackEntry.startIdx) / 2u;

        uint rightIdx;
        if (stackEntry.endIdx - medianIndex - 1 == 0u) {
            rightIdx = INVALID_NODE;
        } else {
            rightIdx = nodeCounter + medianIndex - stackEntry.startIdx + 1;
            stack[stackSize] = StackEntryBuild(medianIndex + 1, stackEntry.endIdx, stackEntry.depth + 1);
            stackSize++;
        }

        uint leftIdx;
        if (medianIndex - stackEntry.startIdx == 0u) {
            leftIdx = INVALID_NODE;
        } else {
            leftIdx = nodeCounter + 1;
            stack[stackSize] = StackEntryBuild(stackEntry.startIdx, medianIndex, stackEntry.depth + 1);
            stackSize++;
        }

        uint leftRightIdx = (leftIdx | rightIdx << 16u);
        nodes[nodeCounter] = KdNode(referenceValues[medianIndex], queryValues[medianIndex], axis, leftRightIdx);
        nodeCounter++;
    }
}

__device__ float findKNearestNeighbors(KdNode* nodes, float point[2], uint e) {
    float distances[k + 1];
    #pragma unroll
    for (int i = 0; i <= k; i++) {
        distances[i] = FLT_MAX;
    }

    uint stack[MAX_STACK_SIZE_KN];
    uint stackSize = 0u;
    uint currNodeIdx = 0u;
    KdNode currNode;
    while (currNodeIdx != INVALID_NODE || stackSize > 0u) {
        while (currNodeIdx != INVALID_NODE) {
            stack[stackSize] = currNodeIdx;
            stackSize++;
            currNode = nodes[currNodeIdx];

            // Descend on side of split planes where the point lies.
            bool isPointOnLeftSide = point[currNode.axis] <= currNode.point[currNode.axis];
            if (isPointOnLeftSide) {
                currNodeIdx = currNode.leftRightIdx & 0x0000FFFFu;
            } else {
                currNodeIdx = (currNode.leftRightIdx & 0xFFFF0000u) >> 16u;
            }
        }

        stackSize--;
        currNodeIdx = stack[stackSize];
        currNode = nodes[currNodeIdx];

        // Compute the distance of this node to the point.
        vec2 diff = make_float2(fabs(point[0] - currNode.point[0]), fabs(point[1] - currNode.point[1]));
        float newDistance = max(diff.x, diff.y);
        if (newDistance < distances[k]) {
            float tempDistance;
            for (int i = 0; i <= k; i++) {
                if (newDistance < distances[i]) {
                    tempDistance = newDistance;
                    newDistance = distances[i];
                    distances[i] = tempDistance;
                }
            }
        }

        // Check whether there could be a closer point on the opposite side.
        bool isPointOnLeftSide = point[currNode.axis] <= currNode.point[currNode.axis];
        if (isPointOnLeftSide && point[currNode.axis] + distances[k] >= currNode.point[currNode.axis]) {
            currNodeIdx = (currNode.leftRightIdx & 0xFFFF0000u) >> 16u;
        } else if (!isPointOnLeftSide && point[currNode.axis] - distances[k] <= currNode.point[currNode.axis]) {
            currNodeIdx = currNode.leftRightIdx & 0x0000FFFFu;
        } else {
            currNodeIdx = INVALID_NODE;
        }
    }
    return distances[k];
}
// ----------------------------------------------------------------------------------


__global__ void mutualInformationKraskov(
        const float* __restrict__ referenceArray, const float* __restrict__ queryArray, float* __restrict__ miArray,
        const uint32_t referenceStride, const uint32_t queryStride,
        const uint32_t batchOffset, const uint32_t batchSize) {
    uint globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x + batchOffset;
    if (globalThreadIdx >= batchSize) {
        return;
    }

    float referenceValues[MEMBER_COUNT];
    float queryValues[MEMBER_COUNT];
    uint offsetReferenceValues = globalThreadIdx * referenceStride;
    uint offsetQueryValues = globalThreadIdx * queryStride;

#ifdef KRASKOV_USE_RANDOM_NOISE
    uint seed = 17u * globalThreadIdx + 240167u;

    // Use Xorshift random numbers with period 2^96-1.
    uvec3 rngState;
    rngState.x = 123456789u ^ seed;
    rngState.y = 362436069u ^ seed;
    rngState.z = 521288629u ^ seed;

    // Optionally add noise.
    for (uint c = 0; c < MEMBER_COUNT; c++) {
        referenceValues[c] = referenceArray[offsetReferenceValues + c] + EPSILON_NOISE * getRandomFloatNorm(rngState);
        queryValues[c] = queryArray[offsetQueryValues + c] + EPSILON_NOISE * getRandomFloatNorm(rngState);
    }
#else
    for (uint c = 0; c < MEMBER_COUNT; c++) {
        referenceValues[c] = referenceArray[offsetReferenceValues + c];
        queryValues[c] = queryArray[offsetQueryValues + c];
    }
#endif

    KdNode nodes[MEMBER_COUNT];
    buildKdTree(nodes, referenceValues, queryValues);

    float point[2];
    float kthNeighborDistances0[MEMBER_COUNT];
    float kthNeighborDistances1[MEMBER_COUNT];
    for (uint c = 0; c < MEMBER_COUNT; c++) {
        point[0] = referenceValues[c];
        point[1] = queryValues[c];
        float val = findKNearestNeighbors(nodes, point, c);
        kthNeighborDistances0[c] = val;
        kthNeighborDistances1[c] = val;
    }

    float a = averageDigamma(kthNeighborDistances0, referenceValues);
    float b = averageDigamma(kthNeighborDistances1, queryValues);
    float c = digamma(k);
    float d = digamma(MEMBER_COUNT);
    //float mi = (-a - b + c + d) / log(base);
    float mi = -a - b + c + d;

    miArray[globalThreadIdx] = mi;
}

}
